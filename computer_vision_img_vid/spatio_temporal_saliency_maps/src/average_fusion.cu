#include "hip/hip_runtime.h"
#include "cpp_includes.h"
#include "fusion_types_header.cuh"
#include "math_functions.h"


texture<unsigned char,1,hipReadModeElementType> tex_dynamic;
texture<unsigned char,1,hipReadModeElementType> tex_static;

int average_fusion_main(unsigned char *maps[], unsigned char *before_fusion[],	const int width, const int height, const int n_maps){
	/* 
		the variable "before_fusion" brings the maps together ready for fusion
	*/
	const int n_pixels = width*height, BLOCKS = (n_pixels+THREADS-1) / THREADS;
	unsigned char *d_static_maps, *d_dynamic_maps, *d_combined;
	hipMalloc((void**)&d_static_maps,sizeof(unsigned char)*n_pixels*n_maps);
	hipMalloc((void**)&d_dynamic_maps,sizeof(unsigned char)*n_pixels*n_maps);
	hipMalloc((void**)&d_combined,sizeof(unsigned char)*n_pixels*n_maps);
    for(int frame=0;frame<n_maps;frame++){
		hipMemcpyAsync(d_static_maps+(frame*n_pixels),before_fusion[frame],sizeof(unsigned char)*n_pixels,hipMemcpyHostToDevice);
		hipMemcpyAsync(d_dynamic_maps+(frame*n_pixels), before_fusion[n_maps + frame],sizeof(unsigned char)*n_pixels,hipMemcpyHostToDevice);
	}

	hipBindTexture(NULL,tex_static,d_static_maps,sizeof(unsigned char)*n_pixels*n_maps);
	hipBindTexture(NULL,tex_dynamic,d_dynamic_maps,sizeof(unsigned char)*n_pixels*n_maps);

	average_kernel<<<BLOCKS,THREADS>>>(d_combined,width,height,n_maps);

	for(int frame=0;frame<n_maps;frame++)
		hipMemcpy(maps[frame], d_combined+(frame*n_pixels),sizeof(unsigned char)*n_pixels,hipMemcpyDeviceToHost);

	hipFree(d_static_maps);
	hipFree(d_dynamic_maps);
	hipFree(d_combined);

	hipUnbindTexture(tex_static);
	hipUnbindTexture(tex_dynamic);
	
	return 0;//exists without a problem
}




__global__ void average_kernel(unsigned char *map, const int width, const int height, const int n_maps){
	unsigned int idx = threadIdx.x + blockIdx.x*blockDim.x;
	int n_pixels = width*height;

	if(idx>=n_pixels) return;//inactive threads

	float static_map;
	float dynamic_map;

	for(int frame=0;frame<n_maps;frame++){//combining static and dynamic maps for each frame
		static_map = (float)tex1Dfetch(tex_static,idx + frame*n_pixels);
		dynamic_map = (float)tex1Dfetch(tex_dynamic,idx + frame*n_pixels);
		map[idx + frame*n_pixels] = (unsigned char)(0.5*(static_map + dynamic_map));
	}
}